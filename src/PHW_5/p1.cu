
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>
#include <iostream>

#define size 1024

__global__ void matrixMul(int *a, int *b, int *c){
	int my_x, my_y;
	my_x = blockIdx.x*blockDim.x + threadIdx.x;	
	my_y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	for(int i = 0 ; i < size; i++)
		local_c += a[my_x * size + i] * b[i * size + my_y];
	
	c[my_x * size + my_y ] = local_c;

}


int main(int argc, char const *argv[])
{
	int n = 1024;
	struct timespec start, stop; 
    double time;
	int* a = (int*)malloc(sizeof(int)*n*n);
	int* b = (int*)malloc(sizeof(int)*n*n);
	int* c = (int*)malloc(sizeof(int)*n*n);

	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			a[i*n + j] = 1;
			b[i*n + j] = 2;
			c[i*n + j] = 0;
		}
	}

	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int)*n*n);
	hipMalloc((void**)&gpu_b, sizeof(int)*n*n);
	hipMalloc((void**)&gpu_c, sizeof(int)*n*n);

	hipMemcpy(gpu_a, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, sizeof(int)*n*n, hipMemcpyHostToDevice);

	dim3 dimGrid(64, 64);
	dim3 dimBlock(16, 16);

	if(clock_gettime(CLOCK_REALTIME, &start) == -1 ) 
	{ 
		perror( "clock gettime" );
	}

	matrixMul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
	hipMemcpy(c, gpu_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
	
	if(clock_gettime(CLOCK_REALTIME, &stop) == -1 ) 
	{ 
		perror( "clock gettime" );
	} 

	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);	
	
	std::cout << c[451*n + 451] << std::endl;
 
	free(a);
	free(b);
	free(c);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);

	return 0;
}