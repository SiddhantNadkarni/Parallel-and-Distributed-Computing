
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>
#include <iostream>
#define size 1024
#define block_size 32


__global__ void matrixMulOptimized(int* a, int* b, int* c)
{


	
	__shared__ float a_share[32][32];
	__shared__ float b_share[32][32];

	int n = 1024;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	int col = blockDim.x*blockIdx.x + threadIdx.x;

	int local_c = 0;
	for (int i = 0; i < 32; ++i)
	{
		a_share[threadIdx.y][threadIdx.x] = a[row*n + i*blockDim.y + threadIdx.x];
		b_share[threadIdx.y][threadIdx.x] = b[(i*blockDim.x + blockIdx.y)*n + col];

		__syncthreads();
		for (int k = 0; k < 32; ++k)
		{
			local_c += a_share[threadIdx.y][k]*b_share[k][threadIdx.x];
		}
		__syncthreads();
		
	}

	c[row*n + col] = local_c;
}


int main(int argc, char const *argv[])
{
	int n = 1024;
	struct timespec start, stop; 
    double time;
	int* a = (int*)malloc(sizeof(int)*n*n);
	int* b = (int*)malloc(sizeof(int)*n*n);
	int* c = (int*)malloc(sizeof(int)*n*n);

	for (int i=0; i<size*size; i++)
	{
		a[i]=1;
		b[i]=2;
		c[i]=0;			
	}

	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int)*n*n);
	hipMalloc((void**)&gpu_b, sizeof(int)*n*n);
	hipMalloc((void**)&gpu_c, sizeof(int)*n*n);

	hipMemcpy(gpu_a, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, sizeof(int)*n*n, hipMemcpyHostToDevice);

	dim3 dimGrid(32,32);
	dim3 dimBlock(32,32);


	if(clock_gettime(CLOCK_REALTIME, &start) == -1 ) 
	{ 
		perror( "clock gettime" );
	}

	matrixMulOptimized<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
	hipMemcpy(c, gpu_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
	
	if(clock_gettime(CLOCK_REALTIME, &stop) == -1 ) 
	{ 
		perror( "clock gettime" );
	} 

	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);	
	
	std::cout << c[451*n + 451] << std::endl;

	free(a);
	free(b);
	free(c);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	return 0;
}